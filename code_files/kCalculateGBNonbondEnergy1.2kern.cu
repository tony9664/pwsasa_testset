#include "hip/hip_runtime.h"
#include "copyright.i"

//---------------------------------------------------------------------------------------------
// AMBER NVIDIA CUDA GPU IMPLEMENTATION: PMEMD VERSION
//
// July 2017, by Scott Le Grand, David S. Cerutti, Daniel J. Mermelstein, Charles Lin, and
//               Ross C. Walker
//---------------------------------------------------------------------------------------------
#include <hip/hip_runtime.h>
#include "gpu.h"
#include "ptxmacros.h"

// Use global instance instead of a local copy 
#include "simulationConst.h"
CSIM_STO simulationConst cSim;

//---------------------------------------------------------------------------------------------
// SetkCalculateGBNonbondEnergy1Sim: this is called by gpuCopyConstants (see gpu.cpp) and is
//                                   jused to port GB constants to the device.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
void SetkCalculateGBNonbondEnergy1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyToSymbol(HIP_SYMBOL(cSim), &gpu->sim, sizeof(simulationConst));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

//---------------------------------------------------------------------------------------------
// SetkCalculateGBNonbondEnergy1Sim: this will download critical GB constants from the device.
//                                   It is not currently called anywhere in the code.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//
// This appears to be a debugging function.
//---------------------------------------------------------------------------------------------
void GetkCalculateGBNonBondEnergy1Sim(gpuContext gpu)
{
    hipError_t status;
    status = hipMemcpyFromSymbol(&gpu->sim, HIP_SYMBOL(cSim), sizeof(simulationConst));
    RTERROR(status, "hipMemcpyToSymbol: SetSim copy to cSim failed");
}

//---------------------------------------------------------------------------------------------
// Kernels for calculating forces and energies for various flavors of GB, even "gas phase"
//---------------------------------------------------------------------------------------------
__global__ void
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
kCalculateGBNonbondForces1_kernel()
#include "kCalculateGBNonbondEnergy1.h"

//---------------------------------------------------------------------------------------------
// Gas phase GB
//---------------------------------------------------------------------------------------------
#define GB_IGB6
__global__ void
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
kCalculateGBNonbondForces1IGB6_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_IGB6

//---------------------------------------------------------------------------------------------
#define GB_ENERGY
__global__
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
void kCalculateGBNonbondEnergy1_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_ENERGY

//---------------------------------------------------------------------------------------------
// Gas phase GB
//---------------------------------------------------------------------------------------------
#define GB_IGB6
#define GB_ENERGY
__global__
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
void kCalculateGBNonbondEnergy1IGB6_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_ENERGY
#undef GB_IGB6

//---------------------------------------------------------------------------------------------
#define GB_MINIMIZATION
__global__ void
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
kCalculateGBNonbondMinimizationForces1_kernel()
#include "kCalculateGBNonbondEnergy1.h"

//---------------------------------------------------------------------------------------------

//---------------------------------------------------------------------------------------------
// Gas phase GB
//---------------------------------------------------------------------------------------------
#define GB_IGB6
__global__ void
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
kCalculateGBNonbondMinimizationForces1IGB6_kernel()
#include "kCalculateGBNonbondEnergy1.h"

//---------------------------------------------------------------------------------------------
#define GB_ENERGY
__global__
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
void kCalculateGBNonbondMinimizationEnergy1IGB6_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_IGB6

//---------------------------------------------------------------------------------------------
__global__
__launch_bounds__(GBNONBONDENERGY1_THREADS_PER_BLOCK,
                  GBNONBONDENERGY1_BLOCKS_MULTIPLIER)
void kCalculateGBNonbondMinimizationEnergy1_kernel()
#include "kCalculateGBNonbondEnergy1.h"
#undef GB_ENERGY
#undef GB_MINIMIZATION

//---------------------------------------------------------------------------------------------
// kCalculateGBNonbondEnergy1InitKernels: what the name says.  Called by gpu_init_ in gpu.cpp.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
void kCalculateGBNonbondEnergy1InitKernels(gpuContext gpu)
{
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy1_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondForces1_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy1IGB6_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondForces1IGB6_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationEnergy1_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationForces1_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationEnergy1IGB6_kernel),
                             hipSharedMemBankSizeEightByte);
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationForces1IGB6_kernel),
                             hipSharedMemBankSizeEightByte);

  if (gpu->sm_version >= SM_3X) {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy1_kernel), hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondForces1_kernel), hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondEnergy1IGB6_kernel), hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondForces1IGB6_kernel), hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationEnergy1_kernel),
                           hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationForces1_kernel),
                           hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationEnergy1IGB6_kernel),
                           hipFuncCachePreferEqual);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kCalculateGBNonbondMinimizationForces1IGB6_kernel),
                           hipFuncCachePreferEqual);
  }
}

//---------------------------------------------------------------------------------------------
// kCalculateGBNonbondForces1: launch the appropriate kernel for computing GB forces.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
extern "C" void kCalculateGBNonbondForces1(gpuContext gpu)
{
  if (gpu->imin == 0) {
    if (gpu->sim.igb != 6) {
      kCalculateGBNonbondForces1_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                          gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
    else {
      kCalculateGBNonbondForces1IGB6_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                              gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
  }
  else {
    if (gpu->sim.igb != 6) {
      kCalculateGBNonbondMinimizationForces1_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                                      gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
    else {
      kCalculateGBNonbondMinimizationForces1IGB6_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                                          gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
  }
  LAUNCHERROR("kCalculateGBNonbondForces1");
}

//---------------------------------------------------------------------------------------------
// kCalculateGBNonbondForces1: launch the appropriate kernel for computing GB energies.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
extern "C" void kCalculateGBNonbondEnergy1(gpuContext gpu)
{
  if (gpu->imin == 0) {
    if (gpu->sim.igb != 6) {
      kCalculateGBNonbondEnergy1_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                          gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
    else {
      kCalculateGBNonbondEnergy1IGB6_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                              gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
  }
  else {
    if (gpu->sim.igb != 6) {
      kCalculateGBNonbondMinimizationEnergy1_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                                      gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
    else {
      kCalculateGBNonbondMinimizationEnergy1IGB6_kernel<<<gpu->GBNonbondEnergy1Blocks,
                                                          gpu->GBNonbondEnergy1ThreadsPerBlock>>>();
    }
  }
  LAUNCHERROR("kCalculateGBNonbondEnergy1");
}

//--------------------------------------------------------------------------------------------
// kReduceMaxsasaEsurf_kernel: Experimenting adding a kernel for reductions of atom loop to add
//                             maxsasa to Esurf from kCalculateGBNonbondEnergy1 kernel 
// -------------------------------------------------------------------------------------------
__global__ void 
__launch_bounds__(REDUCEBUFFER_THREADS_PER_BLOCK, 1)
kReduceMaxsasaEsurf_kernel()
{
  //volatile __shared__ PMEDouble sEsurf[1024];
  volatile __shared__ PMEDouble sEsurf[REDUCEBUFFER_THREADS_PER_BLOCK / GRID]; //pwsasa
  PMEDouble tempes = (PMEDouble)0.0;
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  while (pos < cSim.atoms) {
//#  ifndef use_DPFP
    if ( threadIdx.x == 0 && blockIdx.x == 0 && cSim.surften > 0 ) { //pwsasa correction for maxsasa
       //*cSim.pESurf += llitoulli ( fast_llrintf( ENERGYSCALEF * (PMEFloat)361.108307897 * cSim.surften ));
      //tempes += fast_llrintf( ENERGYSCALEF * (PMEFloat)361.108307897 * cSim.surften ); //surften maxsasa correction
      tempes += (PMEFloat)361.108307897 * cSim.surften; //surften maxsasa correction
       //printf("esurf added once, %d \n", cSim.pgbsa_maxsasa[i]);
      // atomicAdd(cSim.pESurf, llitoulli(tempsurf)); //pwsasa
    }
      //tempes += fast_llrintf( ENERGYSCALEF * (PMEFloat)0.681431329392 * cSim.pgbsa_maxsasa[pos] * cSim.surften );
      tempes += (PMEFloat)0.681431329392 * cSim.pgbsa_maxsasa[pos] * cSim.surften;
      //tempes += fast_llrintf( (PMEFloat)0.681431329392 * cSim.pgbsa_maxsasa[pos] * cSim.surften );
//#  else // use_DPFP
    //if (threadIdx.x == 0 & blockIdx.x == 0){
    //  esurf = (double)((PMEFloat)361.108307897 * cSim.surften) ; //surften maxsasa correction
   // }
   //   esurf = (double)((PMEFloat)0.681431329392 * cSim.pgbsa_maxsasa[i] * cSim.surften) ;
   //   atomicAdd(cSim.pESurf, llitoulli(llrint(esurf * ENERGYSCALE)));
//# endif
      pos += blockDim.x * gridDim.x;
  }

  for (int s=GRID/2; s>=1 ; s/=2){
     tempes += __shfl_down(tempes, s);
  }
  /*tempes  += __SHFL(0xFFFFFFFF, tempes, threadIdx.x ^ 1);
  tempes  += __SHFL(0xFFFFFFFF, tempes, threadIdx.x ^ 2);
  tempes  += __SHFL(0xFFFFFFFF, tempes, threadIdx.x ^ 4);
  tempes  += __SHFL(0xFFFFFFFF, tempes, threadIdx.x ^ 8);
  tempes  += __SHFL(0xFFFFFFFF, tempes, threadIdx.x ^ 16);
  */
  // reduction in blocks
  if (threadIdx.x % GRID == 0) {
     sEsurf[threadIdx.x / GRID] = tempes;
   }
  __syncthreads();

  // red_esurf is in shared memory on a given block
  for (unsigned int s=(REDUCEBUFFER_THREADS_PER_BLOCK/ GRID)/2; s>=1 ; s/=2){
     if (threadIdx.x < s) {
       sEsurf[threadIdx.x] += sEsurf[threadIdx.x+s];
     }
  }

 /*
  // block reduction 
  sEsurf[threadIdx.x]  = tempes;
  sEsurf[threadIdx.x] += sEsurf[threadIdx.x ^ 1];
  sEsurf[threadIdx.x] += sEsurf[threadIdx.x ^ 2];
  sEsurf[threadIdx.x] += sEsurf[threadIdx.x ^ 4];
  sEsurf[threadIdx.x] += sEsurf[threadIdx.x ^ 8];
  sEsurf[threadIdx.x] += sEsurf[threadIdx.x ^ 16];
*/
  if (threadIdx.x  == 0) {
  //if (threadIdx.x  == 0) {
    atomicAdd(cSim.pESurf, llitoulli(llrint(sEsurf[threadIdx.x] * ENERGYSCALEF)));
  } 

}
//pwsasa
extern "C" void kReduceMaxsasaEsurf(gpuContext gpu)
{
  kReduceMaxsasaEsurf_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
  LAUNCHERROR("kReduceMaxsasaEsurf");
}
//pwsasa end


//---------------------------------------------------------------------------------------------
// kReduceGBTemp7_kernel: GB forces reduction kernel
//---------------------------------------------------------------------------------------------
__global__ void
__launch_bounds__(REDUCEBUFFER_THREADS_PER_BLOCK, 1)
kReduceGBTemp7_kernel()
{
  bool bIGB2578 = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  while (pos < cSim.atoms) {
    PMEFloat reff_i       = cSim.pReffSP[pos];
    PMEFloat psi_i        = cSim.pPsi[pos];
    PMEFloat rborn_i      = cSim.pAtomRBorn[pos];
    PMEFloat qi           = cSim.pAtomChargeSP[pos];
    PMEDouble sumdeijda_i = (PMEDouble)cSim.pSumdeijdaAccumulator[pos] * ONEOVERFORCESCALE;

    // Process Temp7 component
    PMEFloat expmkf = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
    PMEFloat dl     = cSim.intdiel_inv - expmkf;
    PMEFloat qi2h   = (PMEFloat)0.50 * qi * qi;
    PMEFloat qid2h  = qi2h * dl;
    sumdeijda_i     = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
    if (cSim.alpb == 0) {

      // egb -= qid2h / reff_i;
    }
    else {

      // egb -= qid2h * (1.0 / reff_i + cSim.one_arad_beta);
      sumdeijda_i *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
    }
    if (bIGB2578) {

      // New onufriev: scale values by alpha-, beta-, gamma- dependent factors later
      PMEFloat thi, thi2;
      if (cSim.igb == 8) {
        PMEFloat alpha = cSim.pgb_alpha[pos];
        PMEFloat gamma = cSim.pgb_gamma[pos];
        PMEFloat beta  = cSim.pgb_beta[pos];
        thi  = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
        thi2 = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i -
                (PMEFloat)2.0 * beta * psi_i) *
               ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
      }
      else {
        thi  = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i -
                     cSim.gb_beta * psi_i) * psi_i);
        thi2 = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i -
                (PMEFloat)2.0 * cSim.gb_beta * psi_i) *
               ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
      }
      sumdeijda_i *= thi2;
    }
    cSim.pTemp7[pos] = sumdeijda_i;
    pos += blockDim.x * gridDim.x;
  }
}

//---------------------------------------------------------------------------------------------
// kReduceGBTemp7: launch the eponymous kernel.  This is invoked in gpu_gb_ene_ (see gpu.cpp)
//                 on all GPUs but the master.
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
extern "C" void kReduceGBTemp7(gpuContext gpu)
{
  kReduceGBTemp7_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
  LAUNCHERROR("kReduceGBTemp7");
}

//---------------------------------------------------------------------------------------------
// kReduceGBTemp7Energy_kernel: GB energies reduction kernel
//---------------------------------------------------------------------------------------------
__global__ void
__launch_bounds__(REDUCEBUFFER_THREADS_PER_BLOCK, 1)
kReduceGBTemp7Energy_kernel()
{
  volatile __shared__ PMEDouble sE[1024];
  bool bIGB2578 = (cSim.igb == 2) || (cSim.igb == 5) || (cSim.igb == 7) || (cSim.igb == 8);
  unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;
  PMEDouble egb = (PMEDouble)0.0;

  while (pos < cSim.atoms) {
    PMEFloat reff_i  = cSim.pReffSP[pos];
    PMEFloat psi_i   = cSim.pPsi[pos];
    PMEFloat rborn_i = cSim.pAtomRBorn[pos];
    PMEFloat qi      = cSim.pAtomChargeSP[pos];
    PMEDouble sumdeijda_i = (PMEDouble)cSim.pSumdeijdaAccumulator[pos] * ONEOVERFORCESCALE;

    // Process Temp7 component
    PMEFloat expmkf = exp(-cSim.gb_kappa * reff_i) * cSim.extdiel_inv;
    PMEFloat dl     = cSim.intdiel_inv - expmkf;
    PMEFloat qi2h   = (PMEFloat)0.50 * qi * qi;
    PMEFloat qid2h  = qi2h * dl;
    sumdeijda_i = -sumdeijda_i + qid2h - cSim.gb_kappa * qi2h * expmkf * reff_i;
    if (cSim.alpb == 0) {
      egb -= qid2h / reff_i;
    }
    else {
      egb -= qid2h * ((PMEFloat)1.0 / reff_i + cSim.one_arad_beta);
      sumdeijda_i *= ((PMEFloat)1.0 + cSim.one_arad_beta * reff_i);
    }
    if (bIGB2578) {

      // New onufriev: we later scale values by alpha-, beta-, and gamma-dependent factor:
      PMEFloat thi, thi2;
      if (cSim.igb == 8) {
        PMEFloat alpha = cSim.pgb_alpha[pos];
        PMEFloat gamma = cSim.pgb_gamma[pos];
        PMEFloat beta  = cSim.pgb_beta[pos];
        thi  = tanh((alpha + gamma * psi_i * psi_i - beta * psi_i) * psi_i);
        thi2 = (alpha + (PMEFloat)3.0 * gamma * psi_i * psi_i -
                (PMEFloat)2.0 * beta * psi_i) *
               ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
      }
      else {
        thi  = tanh((cSim.gb_alpha + cSim.gb_gamma * psi_i * psi_i - cSim.gb_beta * psi_i) *
                    psi_i);
        thi2 = (cSim.gb_alpha + (PMEFloat)3.0 * cSim.gb_gamma * psi_i * psi_i -
                (PMEFloat)2.0 * cSim.gb_beta * psi_i) *
               ((PMEFloat)1.0 - thi * thi) * (rborn_i - cSim.offset) / rborn_i;
      }
      sumdeijda_i *= thi2;
    }
    cSim.pTemp7[pos] = sumdeijda_i;
    pos += blockDim.x * gridDim.x;
  }

  // Reduce Generalized Born energy
  sE[threadIdx.x]  = egb;
  sE[threadIdx.x] += sE[threadIdx.x ^ 1];
  sE[threadIdx.x] += sE[threadIdx.x ^ 2];
  sE[threadIdx.x] += sE[threadIdx.x ^ 4];
  sE[threadIdx.x] += sE[threadIdx.x ^ 8];
  sE[threadIdx.x] += sE[threadIdx.x ^ 16];
  if ((threadIdx.x & GRID_BITS_MASK) == 0) {
    atomicAdd(cSim.pEGB, llitoulli(llrint(sE[threadIdx.x] * ENERGYSCALE)));
  }
}

//---------------------------------------------------------------------------------------------
// kReduceGBTemp7Energy: launch the eponymous kernel
//
// Arguments:
//   gpu: overarching type for storing all parameters, coordinates, and the energy function
//---------------------------------------------------------------------------------------------
extern "C" void kReduceGBTemp7Energy(gpuContext gpu)
{
  kReduceGBTemp7Energy_kernel<<<gpu->blocks, gpu->reduceBufferThreadsPerBlock>>>();
  LAUNCHERROR("kReduceGBTemp7Energy");
}


